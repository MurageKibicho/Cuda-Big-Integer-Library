#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>
#define BASE ((uint64_t)1 << 32)
#define MAX_LIMBS_256 8  // 256 bits
#define MAX_LIMBS_512 16 // 512 bits

struct kibicho_big_256_int_struct
{
	uint32_t digits[MAX_LIMBS_256];
	int sign; //0 is positive, 1 is negative
};
struct kibicho_big_512_int_struct
{
	uint32_t digits[MAX_LIMBS_512];
	int sign;
};

typedef struct kibicho_big_512_int_struct Kib512[1];
typedef struct kibicho_big_256_int_struct Kib256[1];

//Device functions
//memset is not on device. Replace with for loop
__device__ void d_Kib512Zero(Kib512 x)
{
	for(int i = 0; i < MAX_LIMBS_512; ++i)
	{
		x[0].digits[i] = 0;
	}
	x[0].sign = 0;
}

__device__ void d_Kib256Zero(Kib256 x)
{
	for(int i = 0; i < MAX_LIMBS_256; ++i)
	{
		x[0].digits[i] = 0;
	}
	x[0].sign = 0;
}

__device__ int d_Kib256CompareMagnitude(Kib256 a, Kib256 b)
{
	for(int i = MAX_LIMBS_256 - 1; i >= 0; i--)
	{
		if(a[0].digits[i] < b[0].digits[i]) return -1;
		if(a[0].digits[i] > b[0].digits[i]) return 1;
	}
	return 0;
}

__device__ int d_Kib512CompareMagnitude(Kib512 a, Kib512 b)
{
	for(int i = MAX_LIMBS_512 - 1; i >= 0; i--)
	{
		if(a[0].digits[i] < b[0].digits[i]) return -1;
		if(a[0].digits[i] > b[0].digits[i]) return 1;
	}
	return 0;
}

__device__ void d_Kib512AddMagnitudes(Kib512 res, Kib512 a, Kib512 b) 
{
	uint64_t carry = 0;
	for(int i = 0; i < MAX_LIMBS_512; i++)
	{
		uint64_t sum = (uint64_t)a[0].digits[i] + b[0].digits[i] + carry;
		res[0].digits[i] = (uint32_t)(sum & 0xFFFFFFFFULL);
		carry = sum >> 32;
	}
}

__device__ void d_Kib256AddMagnitudes(Kib256 res, Kib256 a, Kib256 b) 
{
	uint64_t carry = 0;
	for(int i = 0; i < MAX_LIMBS_256; i++)
	{
		uint64_t sum = (uint64_t)a[0].digits[i] + b[0].digits[i] + carry;
		res[0].digits[i] = (uint32_t)(sum & 0xFFFFFFFFULL);
		carry = sum >> 32;
	}
}

// Subtract magnitudes (assumes a >= b)
__device__ void d_Kib512SubtractMagnitudes(Kib512 res, Kib512 a, Kib512 b) 
{
	int64_t borrow = 0;
	for(int i = 0; i < MAX_LIMBS_512; i++)
	{
		int64_t diff = (int64_t)a[0].digits[i] - b[0].digits[i] - borrow;
		if(diff < 0)
		{
			diff += BASE;
			borrow = 1;
		}
		else
		{
			borrow = 0;
		}
		res[0].digits[i] = (uint32_t)diff;
	}
}

// Subtract magnitudes (assumes a >= b)
__device__ void d_Kib256SubtractMagnitudes(Kib256 res, Kib256 a, Kib256 b) 
{
	int64_t borrow = 0;
	for(int i = 0; i < MAX_LIMBS_256; i++)
	{
		int64_t diff = (int64_t)a[0].digits[i] - b[0].digits[i] - borrow;
		if(diff < 0)
		{
			diff += BASE;
			borrow = 1;
		}
		else
		{
			borrow = 0;
		}
		res[0].digits[i] = (uint32_t)diff;
	}
}

__device__ void d_Kib512Add(Kib512 res, Kib512 a, Kib512 b) 
{
	if(a[0].sign == b[0].sign)
	{
		d_Kib512AddMagnitudes(res, a, b);
		res[0].sign = a[0].sign;
	}
	else
	{
		int cmp = d_Kib512CompareMagnitude(a, b);
		if(cmp == 0)
		{
			d_Kib512Zero(res);
		}
		else if(cmp > 0)
		{
			d_Kib512SubtractMagnitudes(res, a, b);
			res[0].sign = a[0].sign;
		}
		else
		{
			d_Kib512SubtractMagnitudes(res, b, a);
			res[0].sign = b[0].sign;
		}
	}
}

__device__ void d_Kib256Add(Kib256 res, Kib256 a, Kib256 b) 
{
	if(a[0].sign == b[0].sign)
	{
		d_Kib256AddMagnitudes(res, a, b);
		res[0].sign = a[0].sign;
	}
	else
	{
		int cmp = d_Kib256CompareMagnitude(a, b);
		if(cmp == 0)
		{
			d_Kib256Zero(res);
		}
		else if(cmp > 0)
		{
			d_Kib256SubtractMagnitudes(res, a, b);
			res[0].sign = a[0].sign;
		}
		else
		{
			d_Kib256SubtractMagnitudes(res, b, a);
			res[0].sign = b[0].sign;
		}
	}
}

__device__ void d_Kib256Mul(Kib512 res, Kib256 a, Kib256 b)
{
	d_Kib512Zero(res);
	for(int i = 0; i < MAX_LIMBS_256; i++)
	{
		uint64_t carry = 0;
		for(int j = 0; j < MAX_LIMBS_256; j++)
		{
			uint64_t sum = (uint64_t)a[0].digits[i] * b[0].digits[j] + res[0].digits[i + j] + carry;
			res[0].digits[i + j] = (uint32_t)(sum & 0xFFFFFFFF);
			carry = sum >> 32;
		}
		res[0].digits[i + MAX_LIMBS_256] = (uint32_t)carry;
	}
	//Added sign by XOR
	res[0].sign = a[0].sign ^ b[0].sign;
}

//Kernel
__global__ void kib256MulKernel(Kib512* results, Kib256* inputsA, Kib256* inputsB, int count)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < count)
	{
		d_Kib256Mul(results[idx], inputsA[idx], inputsB[idx]);
	}
}

//Host functions
hipError_t kib256MulBatch(Kib512* d_results, Kib256* d_inputsA, Kib256* d_inputsB, int count)
{
	int threadsPerBlock = 256;
	int blocksPerGrid = (count + threadsPerBlock - 1) / threadsPerBlock;

	kib256MulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_results, d_inputsA, d_inputsB, count);
	return hipGetLastError();
}
hipError_t allocateKib256Array(Kib256** d_array, int count)
{
	return hipMalloc(d_array, count * sizeof(Kib256));
}

hipError_t allocateKib512Array(Kib512** d_array, int count)
{
	return hipMalloc(d_array, count * sizeof(Kib512));
}

hipError_t copyToDeviceKib256(Kib256* d_dest, Kib256* h_src, int count)
{
	return hipMemcpy(d_dest, h_src, count * sizeof(Kib256), hipMemcpyHostToDevice);
}

hipError_t copyFromDeviceKib512(Kib512* h_dest, Kib512* d_src, int count)
{
	return hipMemcpy(h_dest, d_src, count * sizeof(Kib512), hipMemcpyDeviceToHost);
}

//CPU functions
//Rand DNE on CUDA
void Kib512Zero(Kib512 x)
{
	memset(x[0].digits, 0, sizeof(x[0].digits));
	x[0].sign = 0;
}

void Kib256Zero(Kib256 x)
{
	memset(x[0].digits, 0, sizeof(x[0].digits));
	x[0].sign = 0;
}

void Kib512PrintHex(Kib512 a)
{
	int started = 0;
	for (int i = MAX_LIMBS_512 - 1; i >= 0; i--)
	{
		if (!started && a[0].digits[i] == 0) continue;
		if (!started){printf("%x", a[0].digits[i]);started = 1;}
		else{printf("%08x", a[0].digits[i]);}
	}
	if(!started)printf("0");
}

void Kib256PrintHex(Kib256 a)
{
	int started = 0;
	for (int i = MAX_LIMBS_256 - 1; i >= 0; i--)
	{
		if (!started && a[0].digits[i] == 0) continue;
		if (!started){printf("%x", a[0].digits[i]);started = 1;}
		else{printf("%08x", a[0].digits[i]);}
	}
	if(!started)printf("0");
}


void Kib512Random(Kib512 a)
{
	for(int i = 0; i < MAX_LIMBS_512; i++)
	{
		a[0].digits[i] = (uint32_t)rand();
	}
	a[0].sign = rand() % 2;
}

void Kib256Random(Kib256 a)
{
	for(int i = 0; i < MAX_LIMBS_256; i++)
	{
		a[0].digits[i] = (uint32_t)rand();
	}
	a[0].sign = 0;
}

int main()
{
	const int NUM_OPERATIONS = 1000000;
	hipEvent_t start, stop;
	float milliseconds = 0;

	// Create CUDA events for timing
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Allocate host memory
	Kib256* h_inputsA = (Kib256*)malloc(NUM_OPERATIONS * sizeof(Kib256));
	Kib256* h_inputsB = (Kib256*)malloc(NUM_OPERATIONS * sizeof(Kib256));
	Kib512* h_results = (Kib512*)malloc(NUM_OPERATIONS * sizeof(Kib512));

	// Initialize with random data
	for(int i = 0; i < NUM_OPERATIONS; i++)
	{
		Kib256Random(h_inputsA[i]);
		Kib256Random(h_inputsB[i]);
	}

	// Allocate device memory
	Kib256* d_inputsA, *d_inputsB;
	Kib512* d_results;

	allocateKib256Array(&d_inputsA, NUM_OPERATIONS);
	allocateKib256Array(&d_inputsB, NUM_OPERATIONS);
	allocateKib512Array(&d_results, NUM_OPERATIONS);

	// Copy data to device
	copyToDeviceKib256(d_inputsA, h_inputsA, NUM_OPERATIONS);
	copyToDeviceKib256(d_inputsB, h_inputsB, NUM_OPERATIONS);

	// Start timing (includes kernel launch + GPU execution)
	hipEventRecord(start);

	// Launch kernel
	kib256MulBatch(d_results, d_inputsA, d_inputsB, NUM_OPERATIONS);

	// Wait for kernel to complete and stop timing
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	// Calculate elapsed time
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("GPU execution time: %.3f ms\n", milliseconds);
	printf("Operations per second: %.0f\n", NUM_OPERATIONS / (milliseconds / 1000.0f));

	// Copy results back
	copyFromDeviceKib512(h_results, d_results, NUM_OPERATIONS);

	// Cleanup
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(d_inputsA);
	hipFree(d_inputsB);
	hipFree(d_results);
	free(h_inputsA);
	free(h_inputsB);
	free(h_results);

	return 0;
}


